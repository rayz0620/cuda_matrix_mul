//
// Created by rayz on 2019/8/13.
//

#include "matrix_mul.h"
#include "matrix_mul_kernel.h"
#include <hip/hip_runtime.h>

void MatrixMulVanilla(float *mHost, float *nHost, float *pHost, size_t width) {
    float *mDev, *nDev, *pDev;

    size_t matSize = width * width * sizeof(float);

    hipMalloc(&mDev, matSize);
    hipMalloc(&nDev, matSize);
    hipMalloc(&pDev, matSize);

    hipMemcpy(mDev, mHost, matSize, hipMemcpyHostToDevice);
    hipMemcpy(nDev, nHost, matSize, hipMemcpyHostToDevice);

    dim3 gridDim(1, 1);
    dim3 blockDim(width, width);

    matrix_mul_vanilla << < gridDim, blockDim >> > (nDev, mDev, pDev, width);

    hipMemcpy(pHost, pDev, matSize, hipMemcpyDeviceToHost);

    hipFree(pDev);
    hipFree(nDev);
    hipFree(mDev);
}
