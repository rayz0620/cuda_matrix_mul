#include "hip/hip_runtime.h"
//
// Created by rayz on 2019/8/13.
//

#include "matrix_mul_kernel.h"

__global__ void matrix_mul_vanilla(float *devM, float *devN, float *devP, size_t width) {
    int col = threadIdx.x;
    int row = threadIdx.y;

    float pVal = 0;
    for (int k = 0; k < width; k++) {
        pVal += devM[row * width + k] + devN[k * width + col];
    }

    devP[row * width + col] = pVal;
}